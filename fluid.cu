#include "hip/hip_runtime.h"
// Calculate the density contribution of p on q and q on p
__global__ void calculate_density(fluid_particle **fluid_particle_pointers, param *params)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;

    int num_fluid, grid_x, grid_y, bucket_index;
    uint start_index, end_index;
    fluid_particle *p, *q;
    float p_x, p_y, ratio, QmP_x, QmP_y, OmR2;


    num_fluid = params->number_fluid_particles_local + params->number_halo_particles;

    if(i > num_fluid);
        return;

    p = fluid_particle_pointers[i];
    p_x = p->x;
    p_y = p->y;

    // Calculate coordinates within bucket grid
    grid_x = floor(p_x/spacing);
    grid_y = floor(p_y/spacing);

    // Go through neighboring buckets
    for(int dy=-1; dy<=1; dy++) {
        for(int dx=-1; dx<=1; dx++) {

            // If the neighbor bucket is outside of the grid we don't process it
            if ( grid_y+dy < 0 || grid_x+dx < 0 || (grid_x+dx) >= params->grid_size_x || (grid_y+dy) >= params->grid_size_y)
                continue;

             // Linear hash index for bucket
             bucket_index = (grid_y+dy) *params->grid_size_x + grid_x+dx;

             // Start index for hash value of current neighbor bucket
             start_index = start_indexes[bucket_index];

             // If neighbor bucket is not empty
             if (start_index != 0xffffffff)
             {
                end_index = end_indexes[bucket_index];

                for(int j=start_index; j<end_index; j++)
                {
                    q = fluid_particle_pointers[particle_ids[j]];

                    QmP_x = (q->x-p_x);
                    QmP_y = (q->y-p_y);
                    r = sqrt(QmP_x*QmP_x + QmP_y*QmP_y);

                    r_recip = 1.0f/r;
                    ratio = r*h_recip;

                    OmR2 = (1.0f-ratio)*(1.0f-ratio); // (one - r)^2

                    if(ratio < 1.0f) {
                        p->density += OmR2;
                        p->density_near += OmR2*(1.0f-ratio);
                    }
                }
            }
        }
    }
}

__device__ void boundaryConditions(fluid_particle *p, AABB_t *boundary, param *params)
{
    float center_x = params->tunable_params.mover_center_x;
    float center_y = params->tunable_params.mover_center_y;

    // Boundary condition for sphere mover
    if(params->tunable_params.mover_type == SPHERE_MOVER)
    {
        // Sphere width == height
        float radius = params->tunable_params.mover_width*0.5f;
        float norm_x;
        float norm_y;

        // Both circle tests can be combined if no impulse is used
        // Test if inside of circle
        float d;
        float d2 = (p->x - center_x)*(p->x - center_x) + (p->y - center_y)*(p->y - center_y);
        if(d2 <= radius*radius && d2 > 0.0f) {
            d = sqrt(d2);
            norm_x = (center_x-p->x)/d;
            norm_y = (center_y-p->y)/d;

            // With no collision impulse we can handle penetration here
            float pen_dist = radius - d;
            p->x -= pen_dist * norm_x;
            p->y -= pen_dist * norm_y;
        }

    }

    // Make sure object is not outside boundary
    // The particle must not be equal to boundary max or hash potentially won't pick it up
    // as the particle will in the 'next' after last bin
    if(p->x < boundary->min_x) {
        p->x = boundary->min_x;
    }
    else if(p->x > boundary->max_x){
        p->x = boundary->max_x-0.001f;
    }
    if(p->y <  boundary->min_y) {
        p->y = boundary->min_y;
    }
    else if(p->y > boundary->max_y){
        p->y = boundary->max_y-0.001f;
    }
}

__device__ void checkVelocity(float *v_x, float *v_y)
{
    const float v_max = 5.0f;

    if(*v_x > v_max)
        *v_x = v_max;
    else if(*v_x < -v_max)
        *v_x = -v_max;
    if(*v_y > v_max)
        *v_y = v_max;
    else if(*v_y < -v_max)
        *v_y = -v_max;
}

__device__ unsigned int hash_val(float x, float y, neighbor_grid_t *grid, param *params)
{
    float spacing = params->grid_spacing;
    float size_x  = params->grid_size_x;

    // Calculate grid coordinates
    unsigned int grid_x,grid_y;
    grid_x = floor(x/spacing);
    grid_y = floor(y/spacing);

    unsigned int grid_position = (grid_y * size_x + grid_x);

    return grid_position;
}

// The following kernel is modified from CUDA SDK particles example
// rearrange particle data into sorted order, and find the start of each cell
// in the sorted hash array
__global__ void find_cell_start(uint   *start_indexes,        // output: cell start index
                                  uint   *end_indexes,          // output: cell end index
                                  uint   *hash_values, // input: sorted grid hashes
                                  uint   *particle_ids,// input: sorted particle indices
                                  uint    numParticles)
{
    extern __shared__ uint sharedHash[];    // blockSize + 1 elements
    uint index = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;

    uint hash;

    // handle case when no. of particles not multiple of block size
    if (index < numParticles)
    {
        hash = gridParticleHash[index];

        // Load hash data into shared memory so that we can look
        // at neighboring particle's hash value without loading
        // two hash values per thread
        sharedHash[threadIdx.x+1] = hash;

        if (index > 0 && threadIdx.x == 0)
        {
            // first thread in block must load neighbor particle hash
            sharedHash[0] = gridParticleHash[index-1];
        }
    }

    __syncthreads();

    if (index < numParticles)
    {
        // If this particle has a different cell index to the previous
        // particle then it must be the first particle in the cell,
        // so store the index of this particle in the cell.
        // As it isn't the first particle, it must also be the cell end of
        // the previous particle's cell

        if (index == 0 || hash != sharedHash[threadIdx.x])
        {
            start_indexes[hash] = index;

            if (index > 0)
                end_indexes[sharedHash[threadIdx.x]] = index;
        }

        if (index == numParticles - 1)
        {
            end_indexes[hash] = index + 1;
        }
    }

    // Potentially could allocate new_particles_array and reorder as in example
}

__global__ void calculate_hash(fluid_particle **fluid_particle_pointers, uint *hash_values, uint *particle_ids, param *params)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    fluid_particle *p;

    if(i < params->number_fluid_particles_local + params->number_halo_particles )
    {
        p = fluid_particle_pointers[i];
        hash_values[i] =  hash_val(p->x, p->y, params);
        particle_ids[i] = i;
    }
}

__global__ void apply_gravity(fluid_particle **fluid_particle_pointers, param *params)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    fluid_particle *p;
    float dt = params->tunable_params.time_step;
    float g = -params->tunable_params.g;

    if(i < params->number_fluid_particles_local + params->number_halo_particles)
    {
        p = fluid_particle_pointers[i];
        p->v_y += g*dt;

        // Zero out density as well
        p->density = 0.0f;
        p->density_near = 0.0f;
    }
}

__global__ void viscosity_impluses(fluid_particle **fluid_particle_pointers, uint *particle_ids, uint *start_indexes, uint *end_indexes, param *params)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;

    int num_fluid, grid_x, grid_y, bucket_index;
    uint start_index, end_index;
    fluid_particle *p, *q;
    float r, r_recip, ratio, u, imp, imp_x, imp_y;
    float p_x, p_y;
    float QmP_x, QmP_y;
    float h_recip, sigma, beta, dt;

    num_fluid = params->number_fluid_particles_local;
    h_recip = 1.0f/params->tunable_params.smoothing_radius;
    sigma = params->tunable_params.sigma;
    beta = params->tunable_params.beta;
    dt = params->tunable_params.time_step;

    if(i > num_fluid);
        return;

    p = fluid_particle_pointers[i];
    p_x = p->x;
    p_y = p->y;

    // Calculate coordinates within bucket grid
    grid_x = floor(p_x/spacing);
    grid_y = floor(p_y/spacing);

    // Go through neighboring buckets
    for(int dy=-1; dy<=1; dy++) {
        for(int dx=-1; dx<=1; dx++) {

            // If the neighbor bucket is outside of the grid we don't process it
            if ( grid_y+dy < 0 || grid_x+dx < 0 || (grid_x+dx) >= params->grid_size_x || (grid_y+dy) >= params->grid_size_y)
                continue;

             // Linear hash index for bucket
             bucket_index = (grid_y+dy) *params->grid_size_x + grid_x+dx;

             // Start index for hash value of current neighbor bucket
             start_index = start_indexes[bucket_index];

             // If neighbor bucket is not empty
             if (start_index != 0xffffffff)
             {
                end_index = end_indexes[bucket_index];

                for(int j=start_index; j<end_index; j++)
                {
                    q = fluid_particle_pointers[particle_ids[j]];

                    // Continue if same particle
                    if (p==q)
                        continue;

                    QmP_x = (q->x-p_x);
                    QmP_y = (q->y-p_y);
                    r = sqrt(QmP_x*QmP_x + QmP_y*QmP_y);

                    r_recip = 1.0f/r;
                    ratio = r*h_recip;

                    //Inward radial velocity
                    u = ((p->v_x-q->v_x)*QmP_x + (p->v_y-q->v_y)*QmP_y)*r_recip;
                    if(u>0.0f && u<=1.0f)
                    {
                        imp = dt * (1-ratio)*(sigma * u + beta * u*u);
                        imp_x = imp*QmP_x*r_recip;
                        imp_y = imp*QmP_y*r_recip;

                        // Not correct to use velocity check but will stop velocity from
                        // blowing up
                        checkVelocity(&imp_x, &imp_y);

                        p->v_x -= imp_x*0.5f;
                        p->v_y -= imp_y*0.5f;
/*
                    if(q->id < num_fluid) {
                        q->v_x += imp_x*0.5f;
                        q->v_y += imp_y*0.5f;
                    }
                    else { // Only apply half of the impulse to halo particles as they are missing "home" contribution
                        q->v_x += imp_x*0.125f;
                        q->v_y += imp_y*0.125f;
                    }
*/
                    }

                 } // End neighbor bucket particle loop  

             } // bucket not empty

        } // end x
    }  // end y

}

__global__ void predict_positions(fluid_particle **fluid_particle_pointers, AABB_t *boundary_global, param *params)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    num_fluid = params->number_fluid_particles_local;
    fluid_particle *p;
    float dt = params->tunable_params.time_step;

    if(i > num_fluid);
        return;
    p = fluid_particle_pointers[i];
    p->x_prev = p->x;
    p->y_prev = p->y;
    p->x += (p->v_x * dt);
    p->y += (p->v_y * dt);

    // Enforce boundary conditions
    boundaryConditions(p, boundary_global, params);
}

__device__ void updateVelocity(fluid_particle *p, param *params)
{
    float dt = params->tunable_params.time_step;
    float v_x, v_y;

    v_x = (p->x-p->x_prev)/dt;
    v_y = (p->y-p->y_prev)/dt;

    checkVelocity(&v_x, &v_y);

    p->v_x = v_x;
    p->v_y = v_y;
}

__global__ void updateVelocities(fluid_particle **fluid_particle_pointers, AABB_t *boundary_global, param *params)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    num_fluid = params->number_fluid_particles_local;

    if(i > num_fluid);
        return;

    fluid_particle *p;
    p = fluid_particle_pointers[i];
    boundaryConditions(p, boundary_global, params);
    updateVelocity(p, params);
}

__global__ void calculate_pressure(fluid_particle **fluid_particle_pointers, param *params)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    num_fluid = params->number_fluid_particles_local + params->number_halo_particles;

    if(i > num_fluid);
        return;

    p = fluid_particle_pointers[i];
    // Compute pressure and near pressure
    p->pressure = k * (p->density - rest_density);
    p->pressure_near = k_near * p->density_near;
}

__global__ void double_density_relaxation(fluid_particle **fluid_particle_pointers, param *params)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int num_fluid = params->number_fluid_particles_local;

    if(i > num_fluid);
        return;

    int bucket_index, start_index, end_index;
    fluid_particle *p, *q;
    neighbor* n;
    float r,ratio,dt,h,h_recip,r_recip,D,D_x,D_y;
    float k, k_near, k_spring, p_pressure, p_pressure_near, rest_density;
    float OmR;

    num_fluid = params->number_fluid_particles_local;
    k = params->tunable_params.k;
    k_near = params->tunable_params.k_near;
    k_spring = params->tunable_params.k_spring;
    h = params->tunable_params.smoothing_radius;
    h_recip = 1.0f/h;
    dt = params->tunable_params.time_step;
    rest_density = params->tunable_params.rest_density;

    // Iterating through the array in reverse reduces biased particle movement
    p = fluid_particle_pointers[i];
    p_pressure = p->pressure;
    p_pressure_near = p->pressure_near;

    // Calculate coordinates within bucket grid
    grid_x = floor(p->x/spacing);
    grid_y = floor(p->y/spacing);

    // Go through neighboring buckets
    for(int dy=-1; dy<=1; dy++) {
        for(int dx=-1; dx<=1; dx++) {

            // If the neighbor bucket is outside of the grid we don't process it
            if ( grid_y+dy < 0 || grid_x+dx < 0 || (grid_x+dx) >= params->grid_size_x || (grid_y+dy) >= params->grid_size_y)
                continue;

             // Linear hash index for bucket
             bucket_index = (grid_y+dy) *params->grid_size_x + grid_x+dx;

             // Start index for hash value of current neighbor bucket
             start_index = start_indexes[bucket_index];

             // If neighbor bucket is not empty
             if (start_index != 0xffffffff)
             {
                end_index = end_indexes[bucket_index];

                for(int j=start_index; j<end_index; j++)
                {
                    q = fluid_particle_pointers[particle_ids[j]];

                    // Continue if same particle
                    if (p==q)
                        continue;

                    r = sqrt((p->x-q->x)*(p->x-q->x) + (p->y-q->y)*(p->y-q->y));
                    r_recip = 1.0f/r;
                    ratio = r*h_recip;
                    OmR = 1.0f - ratio;

                    // Attempt to move clustered particles apart
                    if(r <= 0.000001f) {
                        p->x += 0.000001f;
                        p->y += 0.000001f;
                    }

                    if(ratio < 1.0f && r > 0.0f) {
                        // Updating both neighbor pairs at the same time, slightly different than the paper but quicker
                        // Also the running sum of D for particle p seems to produce more bias/instability so is removed
                        D = dt*dt*((p_pressure+q->pressure)*OmR + (p_pressure_near+q->pressure_near)*OmR*OmR + k_spring*(h-r)*0.5);
                        D_x = D*(q->x-p->x)*r_recip;
                        D_y = D*(q->y-p->y)*r_recip;
/*
                        // Do not move the halo particles full D
                        // Halo particles are missing D from their origin so I believe this is appropriate
                        if(q->id < num_fluid) {
                            q->x += D_x;
                            q->y += D_y;
                         }
                         else { // Move the halo particles only half way to account for other sides missing contribution
                             q->x += D_x*0.125f;
                             q->y += D_y*0.125f;
                         }
*/
                        p->x -= D_x;
                        p->y -= D_y;
                  } // If in ratio
              }
            }
           }
      } 
}

extern "C" __global__ void double_density_relaxation(fluid_particle **fluid_particle_pointers,param *params)
{
    int total_particles = params->number_fluid_particles_local;
    int block_size = 256;
    int num_blocks = ceil(total_particles/(float)block_size);

    double_density_relaxation<<<num_blocks, block_size>>>(fluid_particle_pointers, params)
}

extern "C" __global__ void calculate_pressures(fluid_particle **fluid_particle_pointers, param *params)
{
    int total_particles = params->number_fluid_particles_local + params->number_halo_particles;
    int block_size = 256;
    int num_blocks = ceil(total_particles/(float)block_size);

    calculate_pressure<<<num_blocks, block_size>>>(fluid_particle_pointers, params)

}

extern "C" __global__ void updateVelocities(fluid_particle **fluid_particle_pointers, AABB_t *boundary_global, param *params)
{
    int total_particles = params->number_fluid_particles_local;
    int block_size = 256;
    int num_blocks = ceil(total_particles/(float)block_size);

    updateVelocities<<<num_blocks, block_size>>>(fluid_particle_pointers, boundary_global, params)
}

extern "C" predict_positions(fluid_particle **fluid_particle_pointers, AABB_t *boundary_global, param *params)
{
    int total_particles = params->number_fluid_particles_local;
    int block_size = 256;
    int num_blocks = ceil(total_particles/(float)block_size);

    predict_positions<<<num_blocks, block_size>>>(fluid_particle_pointers, boundary_global, params);
}

extern "C" void hash_particles(fluid_particle **fluid_particle_pointers, uint *hash_values, uint *particle_ids, uint *starts, uint *ends, *params)
{
    int total_particles = params->number_fluid_particles_local + params->number_halo_particles;
    int block_size = 256;
    int num_blocks = ceil(total_particles/(float)block_size);

    // Reset start indexes
    unsigned int length_hash = params->grid_size_x * params->grid_size_y;
    hipMemset(starts, 0xffffffff, length_hash*sizeof(uint));

    // Hash particles
    calculate_hash(fluid_particle_pointers, hash_values, particle_ids, params);  

    // Sort hashed values
    sort_hash<<<num_blocks, block_size>>>(particle_ids, hash_values, params);

    // Find start/end indexes for sorted values
    find_cell_start<<<num_blocks, block_size>>>(starts, ends, hash_values, particle_ids, total_particles);

    // Wait for kernels to complete
    hipDeviceSynchronize();
}

// Use thrust radix sort to sort ()
// Could also use uint2 and leap iterator...
extern "C" void sort_hash(uint *d_particle_ids, uint *d_hash_values, param *params)
{
    int total_particles = params->number_fluid_particles_local + params->number_halo_particles;

    thrust::sort_by_key(thrust::device_ptr<uint>(d_hash_values),
                        thrust::device_ptr<uint>(d_hash_values + total_particles),
                        thrust::device_ptr<uint>(d_particle_ids)
                        );
}

extern "C" void apply_gravity(fluid_particle **fluid_particle_pointers, param *params)
{
    num_blocks = ceil( (params.number_fluid_particles_local + params.number_halo_particles)/(float)threads_per_block );
    apply_gravity<<< num_blocks, threads_per_block >>>(fluid_particle_pointers, &params);
}

extern "C" void calculate_density(fluid_particle **fluid_particle_pointers, param *params)
{
    int total_particles = params->number_fluid_particles_local + params->number_halo_particles;
    int block_size = 256;
    int num_blocks = ceil(total_particles/(float)block_size);

    calculate_density<<<num_blocks, threads_per_block>>>(fluid_particle_pointers, params);
}

extern "C" void viscosity_impluses(fluid_particle **fluid_particle_pointers, uint *particle_ids, uint *start_indexes, uint *end_indexes, param *params)
{
    int total_particles = params->number_fluid_particles_local; //+ params->number_halo_particles;
    viscosity_impluses(fluid_particle_pointers, particle_ids, start_indexes, end_indexes, params);
}
