#include "hip/hip_runtime.h"
#include "thrust/device_ptr.h"
#include "thrust/sort.h"
#include "fluid.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// Calculate the density contribution of p on q and q on p
__global__ void calculate_density(fluid_particle **fluid_particle_pointers, uint *start_indexes, uint *end_indexes, uint *particle_ids, param *params)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;

    int num_fluid, grid_x, grid_y, bucket_index;
    uint start_index, end_index;
    fluid_particle *p, *q;
    float p_x, p_y, ratio, QmP_x, QmP_y, OmR2, r, h_recip;
    h_recip = 1.0f/params->tunable_params.smoothing_radius;

    num_fluid = params->number_fluid_particles_local + params->number_halo_particles;

    if(i >= num_fluid)
        return;

    p = fluid_particle_pointers[i];
    p_x = p->x;
    p_y = p->y;

    float spacing = params->grid_spacing;

    // Calculate coordinates within bucket grid
    grid_x = floor(p_x/spacing);
    grid_y = floor(p_y/spacing);

    // Go through neighboring buckets
    for(int dy=-1; dy<=1; dy++) {
        for(int dx=-1; dx<=1; dx++) {

            // If the neighbor bucket is outside of the grid we don't process it
            if ( grid_y+dy < 0 || grid_x+dx < 0 || (grid_x+dx) >= params->grid_size_x || (grid_y+dy) >= params->grid_size_y)
                continue;

             // Linear hash index for bucket
             bucket_index = (grid_y+dy) *params->grid_size_x + grid_x+dx;

             // Start index for hash value of current neighbor bucket
             start_index = start_indexes[bucket_index];

             // If neighbor bucket is not empty
             if (start_index != 0xffffffff)
             {
                end_index = end_indexes[bucket_index];

                for(int j=start_index; j<end_index; j++)
                {
                    q = fluid_particle_pointers[particle_ids[j]];

                    QmP_x = (q->x-p_x);
                    QmP_y = (q->y-p_y);
                    r = sqrt(QmP_x*QmP_x + QmP_y*QmP_y);

                    ratio = r*h_recip;

                    OmR2 = (1.0f-ratio)*(1.0f-ratio); // (one - r)^2

                    if(ratio < 1.0f) {
                        p->density += OmR2;
                        p->density_near += OmR2*(1.0f-ratio);
                    }
                }
            }
        }
    }
}

__device__ void boundaryConditions(fluid_particle *p, AABB_t *boundary, param *params)
{
    float center_x = params->tunable_params.mover_center_x;
    float center_y = params->tunable_params.mover_center_y;

    // Boundary condition for sphere mover
    if(params->tunable_params.mover_type == SPHERE_MOVER)
    {
        // Sphere width == height
        float radius = params->tunable_params.mover_width*0.5f;
        float norm_x;
        float norm_y;

        // Both circle tests can be combined if no impulse is used
        // Test if inside of circle
        float d;
        float d2 = (p->x - center_x)*(p->x - center_x) + (p->y - center_y)*(p->y - center_y);
        if(d2 <= radius*radius && d2 > 0.0f) {
            d = sqrt(d2);
            norm_x = (center_x-p->x)/d;
            norm_y = (center_y-p->y)/d;

            // With no collision impulse we can handle penetration here
            float pen_dist = radius - d;
            p->x -= pen_dist * norm_x;
            p->y -= pen_dist * norm_y;
        }

    }

    // Make sure object is not outside boundary
    // The particle must not be equal to boundary max or hash potentially won't pick it up
    // as the particle will in the 'next' after last bin
    if(p->x < boundary->min_x) {
        p->x = boundary->min_x;
    }
    else if(p->x > boundary->max_x){
        p->x = boundary->max_x-0.001f;
    }
    if(p->y <  boundary->min_y) {
        p->y = boundary->min_y;
    }
    else if(p->y > boundary->max_y){
        p->y = boundary->max_y-0.001f;
    }
}

__device__ void checkVelocity(float *v_x, float *v_y)
{
    const float v_max = 5.0f;

    if(*v_x > v_max)
        *v_x = v_max;
    else if(*v_x < -v_max)
        *v_x = -v_max;
    if(*v_y > v_max)
        *v_y = v_max;
    else if(*v_y < -v_max)
        *v_y = -v_max;
}

__device__ uint hash_val(float x, float y, param *params)
{
    float spacing = params->grid_spacing;
    float size_x  = params->grid_size_x;

    // Calculate grid coordinates
    uint grid_x,grid_y;
    grid_x = floor(x/spacing);
    grid_y = floor(y/spacing);

    uint grid_position = (grid_y * size_x + grid_x);

    return grid_position;
}

// The following kernel is modified from CUDA SDK particles example
// rearrange particle data into sorted order, and find the start of each cell
// in the sorted hash array
__global__ void find_cell_start(uint   *start_indexes,        // output: cell start index
                                  uint   *end_indexes,          // output: cell end index
                                  uint   *hash_values, // input: sorted grid hashes
                                  uint   *particle_ids,// input: sorted particle indices
                                  uint    numParticles)
{
    extern __shared__ uint sharedHash[];    // blockSize + 1 elements
    uint index = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;

    uint hash;

    // handle case when no. of particles not multiple of block size
    if (index < numParticles)
    {
        hash = hash_values[index];

        // Load hash data into shared memory so that we can look
        // at neighboring particle's hash value without loading
        // two hash values per thread
        sharedHash[threadIdx.x+1] = hash;

        if (index > 0 && threadIdx.x == 0)
        {
            // first thread in block must load neighbor particle hash
            sharedHash[0] = hash_values[index-1];
        }
    }

    __syncthreads();

    if (index < numParticles)
    {
        // If this particle has a different cell index to the previous
        // particle then it must be the first particle in the cell,
        // so store the index of this particle in the cell.
        // As it isn't the first particle, it must also be the cell end of
        // the previous particle's cell

        if (index == 0 || hash != sharedHash[threadIdx.x])
        {
            start_indexes[hash] = index;

            if (index > 0)
                end_indexes[sharedHash[threadIdx.x]] = index;
        }

        if (index == numParticles - 1)
        {
            end_indexes[hash] = index + 1;
        }
    }

    // Potentially could allocate new_particles_array and reorder as in example
}

__global__ void calculate_hash(fluid_particle **fluid_particle_pointers, uint *hash_values, uint *particle_ids, param *params)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    fluid_particle *p;
    int num_fluid = params->number_fluid_particles_local + params->number_halo_particles;

    if(i >= num_fluid)
        return;

    p = fluid_particle_pointers[i];
    hash_values[i] =  hash_val(p->x, p->y, params);
    particle_ids[i] = i;
}

__global__ void apply_gravity(fluid_particle **fluid_particle_pointers, param *params)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;

    fluid_particle *p;
    float dt = params->tunable_params.time_step;
    float g = -params->tunable_params.g;

    int num_fluid = params->number_fluid_particles_local + params->number_halo_particles;

    if(i >= num_fluid)
        return;

    p = fluid_particle_pointers[i];
    p->v_y += g*dt;

    // Zero out density as well
    p->density = 0.0f;
    p->density_near = 0.0f;
}

__global__ void viscosity_impluses(fluid_particle **fluid_particle_pointers, uint *particle_ids, uint *start_indexes, uint *end_indexes, param *params)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;

    int num_fluid, grid_x, grid_y, bucket_index;
    uint start_index, end_index;
    fluid_particle *p, *q;
    float r, r_recip, ratio, u, imp, imp_x, imp_y;
    float p_x, p_y;
    float QmP_x, QmP_y;
    float h_recip, sigma, beta, dt,spacing;

    num_fluid = params->number_fluid_particles_local;
    h_recip = 1.0f/params->tunable_params.smoothing_radius;
    sigma = params->tunable_params.sigma;
    beta = params->tunable_params.beta;
    dt = params->tunable_params.time_step;
    spacing = params->grid_spacing;

    if(i >= num_fluid)
        return;

    p = fluid_particle_pointers[i];
    p_x = p->x;
    p_y = p->y;

    // Calculate coordinates within bucket grid
    grid_x = floor(p_x/spacing);
    grid_y = floor(p_y/spacing);

    // Go through neighboring buckets
    for(int dy=-1; dy<=1; dy++) {
        for(int dx=-1; dx<=1; dx++) {

            // If the neighbor bucket is outside of the grid we don't process it
            if ( grid_y+dy < 0 || grid_x+dx < 0 || (grid_x+dx) >= params->grid_size_x || (grid_y+dy) >= params->grid_size_y)
                continue;

             // Linear hash index for bucket
             bucket_index = (grid_y+dy) *params->grid_size_x + grid_x+dx;

             // Start index for hash value of current neighbor bucket
             start_index = start_indexes[bucket_index];

             // If neighbor bucket is not empty
             if (start_index != 0xffffffff)
             {
                end_index = end_indexes[bucket_index];

                for(int j=start_index; j<end_index; j++)
                {
                    q = fluid_particle_pointers[particle_ids[j]];

                    // Continue if same particle
                    if (p==q)
                        continue;

                    QmP_x = (q->x-p_x);
                    QmP_y = (q->y-p_y);
                    r = sqrt(QmP_x*QmP_x + QmP_y*QmP_y);

                    r_recip = 1.0f/r;
                    ratio = r*h_recip;

                    //Inward radial velocity
                    u = ((p->v_x-q->v_x)*QmP_x + (p->v_y-q->v_y)*QmP_y)*r_recip;

                    if(u>0.0f && u<=1.0f)
                    {
                        imp = dt * (1-ratio)*(sigma * u + beta * u*u);
                        imp_x = imp*QmP_x*r_recip;
                        imp_y = imp*QmP_y*r_recip;

                        // Not correct to use velocity check but will stop velocity from
                        // blowing up
                        checkVelocity(&imp_x, &imp_y);

//                        p->v_x -= imp_x*0.5f;
//                        p->v_y -= imp_y*0.5f;
                    }
                 } // End neighbor bucket particle loop  

             } // bucket not empty

        } // end x
    }  // end y

}

__global__ void predict_positions(fluid_particle **fluid_particle_pointers, AABB_t *boundary_global, param *params)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int num_fluid = params->number_fluid_particles_local;
    fluid_particle *p;
    float dt = params->tunable_params.time_step;

    if(i >= num_fluid)
        return;
    p = fluid_particle_pointers[i];
    p->x_prev = p->x;
    p->y_prev = p->y;
    p->x += (p->v_x * dt);
    p->y += (p->v_y * dt);

    // Enforce boundary conditions
    boundaryConditions(p, boundary_global, params);
}

__device__ void updateVelocity(fluid_particle *p, param *params)
{
    float dt = params->tunable_params.time_step;
    float v_x, v_y;

    v_x = (p->x-p->x_prev)/dt;
    v_y = (p->y-p->y_prev)/dt;

    checkVelocity(&v_x, &v_y);

    p->v_x = v_x;
    p->v_y = v_y;
}

__global__ void updateVelocities(fluid_particle **fluid_particle_pointers, AABB_t *boundary_global, param *params)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int num_fluid = params->number_fluid_particles_local;

    if(i >= num_fluid)
        return;

    fluid_particle *p;
    p = fluid_particle_pointers[i];
    boundaryConditions(p, boundary_global, params);
    updateVelocity(p, params);
}

__global__ void calculate_pressure(fluid_particle **fluid_particle_pointers, param *params)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int num_fluid = params->number_fluid_particles_local + params->number_halo_particles;

    float k, k_near, rest_density;

    k = params->tunable_params.k;
    k_near = params->tunable_params.k_near;
    rest_density = params->tunable_params.rest_density;

    fluid_particle *p;

    if(i >= num_fluid)
        return;

    p = fluid_particle_pointers[i];
    // Compute pressure and near pressure
    p->pressure = k * (p->density - rest_density);
    p->pressure_near = k_near * p->density_near;
}

__global__ void double_density_relaxation(fluid_particle **fluid_particle_pointers, uint *particle_ids, uint *start_indexes, uint *end_indexes, param *params)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int num_fluid = params->number_fluid_particles_local;

    if(i >= num_fluid)
        return;

    int bucket_index, start_index, end_index;
    fluid_particle *p, *q;
    float r,ratio,dt,h,h_recip,r_recip,D,D_x,D_y;
    float k_spring, p_pressure, p_pressure_near;
    float OmR, grid_x, grid_y;

    num_fluid = params->number_fluid_particles_local;
    k_spring = params->tunable_params.k_spring;
    h = params->tunable_params.smoothing_radius;
    h_recip = 1.0f/h;
    dt = params->tunable_params.time_step;

    // Iterating through the array in reverse reduces biased particle movement
    p = fluid_particle_pointers[i];
    p_pressure = p->pressure;
    p_pressure_near = p->pressure_near;

    float spacing = params->grid_spacing;

    // Calculate coordinates within bucket grid
    grid_x = floor(p->x/spacing);
    grid_y = floor(p->y/spacing);

    // Go through neighboring buckets
    for(int dy=-1; dy<=1; dy++) {
        for(int dx=-1; dx<=1; dx++) {

            // If the neighbor bucket is outside of the grid we don't process it
            if ( grid_y+dy < 0 || grid_x+dx < 0 || (grid_x+dx) >= params->grid_size_x || (grid_y+dy) >= params->grid_size_y)
                continue;

             // Linear hash index for bucket
             bucket_index = (grid_y+dy) *params->grid_size_x + grid_x+dx;

             // Start index for hash value of current neighbor bucket
             start_index = start_indexes[bucket_index];

             // If neighbor bucket is not empty
             if (start_index != 0xffffffff)
             {
                end_index = end_indexes[bucket_index];

                for(int j=start_index; j<end_index; j++)
                {
                    q = fluid_particle_pointers[particle_ids[j]];

                    // Continue if same particle
                    if (p==q)
                        continue;

                    r = sqrt((p->x-q->x)*(p->x-q->x) + (p->y-q->y)*(p->y-q->y));
                    r_recip = 1.0f/r;
                    ratio = r*h_recip;
                    OmR = 1.0f - ratio;

                    // Attempt to move clustered particles apart
                    if(r <= 0.000001f) {
                        p->x += 0.000001f;
                        p->y += 0.000001f;
                    }

                    if(ratio < 1.0f && r > 0.0f) {
                        // Updating both neighbor pairs at the same time, slightly different than the paper but quicker
                        // Also the running sum of D for particle p seems to produce more bias/instability so is removed
                        D = dt*dt*((p_pressure+q->pressure)*OmR + (p_pressure_near+q->pressure_near)*OmR*OmR + k_spring*(h-r)*0.5);
                        D_x = D*(q->x-p->x)*r_recip;
                        D_y = D*(q->y-p->y)*r_recip;

                        p->x -= D_x;
                        p->y -= D_y;
                  } // If in ratio
              }
            }
           }
      } 
}

extern "C" void double_density_relaxation_gpu(fluid_particle **fluid_particle_pointers, uint *particle_ids, uint *start_indexes, uint *end_indexes, param *params)
{
    int total_particles = params->number_fluid_particles_local;
    int block_size = 256;
    int num_blocks = ceil(total_particles/(float)block_size);

    double_density_relaxation<<<num_blocks, block_size>>>(fluid_particle_pointers, particle_ids, start_indexes, end_indexes, params);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

}

extern "C" void calculate_pressures_gpu(fluid_particle **fluid_particle_pointers, param *params)
{
    int total_particles = params->number_fluid_particles_local + params->number_halo_particles;
    int block_size = 256;
    int num_blocks = ceil(total_particles/(float)block_size);

    calculate_pressure<<<num_blocks, block_size>>>(fluid_particle_pointers, params);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );


}

extern "C" void updateVelocities_gpu(fluid_particle **fluid_particle_pointers, AABB_t *boundary_global, param *params)
{
    int total_particles = params->number_fluid_particles_local;
    int block_size = 256;
    int num_blocks = ceil(total_particles/(float)block_size);

    updateVelocities<<<num_blocks, block_size>>>(fluid_particle_pointers, boundary_global, params);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

}

extern "C" void predict_positions_gpu(fluid_particle **fluid_particle_pointers, AABB_t *boundary_global, param *params)
{
    int total_particles = params->number_fluid_particles_local;
    int block_size = 256;
    int num_blocks = ceil(total_particles/(float)block_size);

    predict_positions<<<num_blocks, block_size>>>(fluid_particle_pointers, boundary_global, params);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

}

// Use thrust radix sort to sort ()
// Could also use uint2 and leap iterator...
extern "C" void sort_hash_gpu(uint *d_particle_ids, uint *d_hash_values, param *params)
{
    int total_particles = params->number_fluid_particles_local + params->number_halo_particles;

    thrust::device_ptr<uint> keys(d_hash_values);
    thrust::device_ptr<uint> values(d_particle_ids);

    thrust::sort_by_key(keys, keys+total_particles, values);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
}

extern "C" void hash_particles_gpu(fluid_particle **fluid_particle_pointers, uint *hash_values, uint *particle_ids, uint *starts, uint *ends, param *params)
{
    int total_particles = params->number_fluid_particles_local + params->number_halo_particles;
    int block_size = 256;
    int num_blocks = ceil(total_particles/(float)block_size);

    // Reset start indexes
    unsigned int length_hash = params->grid_size_x * params->grid_size_y;
    hipMemset(starts, 0xffffffff, length_hash*sizeof(uint));

    // Hash particles
    calculate_hash<<<num_blocks, block_size>>>(fluid_particle_pointers, hash_values, particle_ids, params);  

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    // Sort hashed values
    sort_hash_gpu(particle_ids, hash_values, params);

    // Find start/end indexes for sorted values
    uint smem_size = sizeof(uint)*(block_size+1);
    num_blocks = ceil(length_hash/(float)block_size);
    find_cell_start<<<num_blocks, block_size, smem_size>>>(starts, ends, hash_values, particle_ids, total_particles);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

}

extern "C" void apply_gravity_gpu(fluid_particle **fluid_particle_pointers, param *params)
{
    int total_particles = params->number_fluid_particles_local + params->number_halo_particles;
    int block_size = 256;
    int num_blocks = ceil(total_particles/(float)block_size);

    apply_gravity<<< num_blocks, block_size >>>(fluid_particle_pointers, params);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
}

extern "C" void calculate_density_gpu(fluid_particle **fluid_particle_pointers, uint *start_indexes, uint *end_indexes, uint *particle_ids, param *params)
{
    int total_particles = params->number_fluid_particles_local + params->number_halo_particles;
    int block_size = 256;
    int num_blocks = ceil(total_particles/(float)block_size);

    calculate_density<<<num_blocks, block_size>>>(fluid_particle_pointers, start_indexes, end_indexes, particle_ids, params);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

}

extern "C" void viscosity_impluses_gpu(fluid_particle **fluid_particle_pointers, uint *particle_ids, uint *start_indexes, uint *end_indexes, param *params)
{
    int total_particles = params->number_fluid_particles_local;
    int block_size = 256;
    int num_blocks = ceil(total_particles/(float)block_size);

    viscosity_impluses<<<num_blocks, block_size>>>(fluid_particle_pointers, particle_ids, start_indexes, end_indexes, params);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

}
